#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <hip/hip_fp16.h>

#define ROWM 1
#define COLM 100
#define COLN 100

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)



typedef __half mytype;
// Pi = Mi x Ni
// pr = ROWM
// pc = COLN
// mc = COLM

void GPU_Multi(mytype **M, mytype **N, mytype **P
  , size_t pr, size_t pc, size_t mc
  , size_t num_mat, mytype alpha, mytype beta)
{
    #define NUM_MAT  10000
    mytype *devM[num_mat];
    mytype *devN[num_mat];
    mytype *devP[num_mat];
    size_t p_size =sizeof(mytype) *pr*pc;
    size_t m_size =sizeof(mytype) *pr*mc;
    size_t n_size =sizeof(mytype) *mc*pc;
    const mytype **d_Marray, **d_Narray;
    mytype **d_Parray;
    hipblasHandle_t myhandle;
    hipblasStatus_t cublas_result;
 

 
    for(int i = 0 ; i < num_mat; i ++ )
    {
        hipMalloc((void**)&devM[ i ], m_size );
        hipMalloc((void**)&devN[ i ], n_size );
        hipMalloc((void**)&devP[ i ], p_size );
    }
    hipMalloc((void**)&d_Marray, num_mat*sizeof(mytype *));
    hipMalloc((void**)&d_Narray, num_mat*sizeof(mytype *));
    hipMalloc((void**)&d_Parray, num_mat*sizeof(mytype *));
    cudaCheckErrors("hipMalloc fail");
    for(int i = 0 ; i < num_mat; i ++ ) {

        hipMemcpy(devM[i], M[i], m_size , hipMemcpyHostToDevice);
        hipMemcpy(devN[i], N[i], n_size , hipMemcpyHostToDevice);
        hipMemcpy(devP[i], P[i], p_size , hipMemcpyHostToDevice);
    }
    hipMemcpy(d_Marray, devM, num_mat*sizeof(mytype *), hipMemcpyHostToDevice);
    hipMemcpy(d_Narray, devN, num_mat*sizeof(mytype *), hipMemcpyHostToDevice);
    hipMemcpy(d_Parray, devP, num_mat*sizeof(mytype *), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    cublas_result = hipblasCreate(&myhandle);
    assert(cublas_result == HIPBLAS_STATUS_SUCCESS);
    // change to    cublasDgemmBatched for double
    for (int i=0 ; i<50000 ; i++)
    {  
     cublas_result = hipblasHgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N
      , pr, pc, mc
      , &alpha, d_Marray, pr, d_Narray, mc
      , &beta, d_Parray, pr
      , num_mat);
    }
 
    //std::cout<<cudaElapsedTime<<std::endl;
    assert(cublas_result == HIPBLAS_STATUS_SUCCESS);
    

    for(int i = 0 ; i < num_mat ; i ++ )
    {
        hipMemcpy(P[i], devP[i], p_size, hipMemcpyDeviceToHost);
        hipFree(devM[i]);
        hipFree(devN[i]);
        hipFree(devP[i]);
    }
    hipFree(d_Marray);
    hipFree(d_Narray);
    hipFree(d_Parray);
    cudaCheckErrors("hipMemcpy D2H fail");


}

int main(){
  mytype h_M1[ROWM][COLM], h_M2[ROWM][COLM];
  mytype h_N1[COLM][COLN], h_N2[COLM][COLN];
  mytype h_P1[ROWM][COLN], h_P2[ROWM][COLN];
  mytype *h_Marray[10000], *h_Narray[10000], *h_Parray[10000];
  for (int i = 0; i < ROWM; i++)
    for (int j = 0; j < COLM; j++){
      h_M1[i][j] = 1.0f; h_M2[i][j] = 2.0f;}
  for (int i = 0; i < COLM; i++)
    for (int j = 0; j < COLN; j++){
      h_N1[i][j] = 1.0f; h_N2[i][j] = 1.0f;}
  for (int i = 0; i < ROWM; i++)
    for (int j = 0; j < COLN; j++){
      h_P1[i][j] = 0.0f; h_P2[i][j] = 0.0f;}

  for (int i = 0; i < 10000; i+=2)
  {
   h_Marray[0+i] = &(h_M1[0][0]);
   h_Marray[1+i] = &(h_M2[0][0]);
   h_Narray[0+i] = &(h_N1[0][0]);
   h_Narray[1+i] = &(h_N2[0][0]);
   h_Parray[0+i] = &(h_P1[0][0]);
   h_Parray[1+i] = &(h_P2[0][0]);
  }

  GPU_Multi(h_Marray, h_Narray, h_Parray, ROWM, COLN, COLM, 10000, 1.0f, 1.0f);

  printf("Success!\n");
  return 0;
}
